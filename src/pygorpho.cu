#include "hip/hip_runtime.h"
#include "pygorpho.cuh"

#include <cstring>

#include "general_morph.cuh"
#include "flat_morph.cuh"
#include "flat_linear_morph.cuh"
#include "strel.cuh"
#include "view.cuh"

template <class Ty>
void doGenDilateErode(void *resPtr, const void *volPtr, const void *strelPtr, int3 volSize, int3 strelSize,
    int op, int3 blockSize)
{
    gpho::HostView<Ty> res(static_cast<Ty *>(resPtr), volSize);
    gpho::HostView<const Ty> vol(static_cast<const Ty *>(volPtr), volSize);
    gpho::HostView<const Ty> strel(static_cast<const Ty *>(strelPtr), strelSize);

    if (op == MOP_DILATE) {
        gpho::genDilate(res, vol, strel, blockSize);
    } else if (op == MOP_ERODE) {
        gpho::genErode(res, vol, strel, blockSize);
    } else {
        throw ERR_BAD_MORPH_OP;
    }
}

template <class Ty>
void doFlatDilateErode(void *resPtr, const void *volPtr, const bool *strelPtr, int3 volSize, int3 strelSize,
    int op, int3 blockSize)
{
    gpho::HostView<Ty> res(static_cast<Ty *>(resPtr), volSize);
    gpho::HostView<const Ty> vol(static_cast<const Ty *>(volPtr), volSize);
    gpho::HostView<const bool> strel(strelPtr, strelSize);

    if (op == MOP_DILATE) {
        gpho::flatDilate(res, vol, strel, blockSize);
    } else if (op == MOP_ERODE) {
        gpho::flatErode(res, vol, strel, blockSize);
    } else {
        throw ERR_BAD_MORPH_OP;
    }
}

template <class Ty>
void doFlatLinearDilateErode(void *resPtr, const void *volPtr, const int *lineStepsPtr,
    const int *lineLensPtr, int3 volSize, int numLines, int op, int3 blockSize)
{
    std::vector<gpho::LineSeg> lines;
    lines.reserve(numLines);
    for (int i = 0; i < numLines; ++i) {
        int3 step = make_int3(
            lineStepsPtr[i * 3 + 0],
            lineStepsPtr[i * 3 + 1],
            lineStepsPtr[i * 3 + 2]
        );
        lines.push_back(gpho::LineSeg(step, lineLensPtr[i]));
    }

    gpho::HostView<Ty> res(static_cast<Ty *>(resPtr), volSize);
    gpho::HostView<const Ty> vol(static_cast<const Ty *>(volPtr), volSize);

    if (op == MOP_DILATE) {
        gpho::flatLinearDilateErode<gpho::MORPH_DILATE>(res, vol, lines, blockSize);
    } else if (op == MOP_ERODE) {
        gpho::flatLinearDilateErode<gpho::MORPH_ERODE>(res, vol, lines, blockSize);
    } else {
        throw ERR_BAD_MORPH_OP;
    }
}

gpho::ApproxType getApproxTypeFromCode(int typeCode)
{
    switch (typeCode) {
        case AT_INSIDE:
            return gpho::APPROX_INSIDE;
        case AT_BEST:
            return gpho::APPROX_BEST;
        case AT_OUTSIDE:
            return gpho::APPROX_OUTSIDE;
        default:
            throw ERR_BAD_APPROX_TYPE;
    }
}

#ifdef __cplusplus
extern "C" {
#endif

PYGORPHO_API int pyDilateOp() { return MOP_DILATE; };
PYGORPHO_API int pyErodeOp() { return MOP_ERODE; };

PYGORPHO_API int pyGetDeviceCount()
{
    int ndev = 0;
    hipGetDeviceCount(&ndev); // It seems like we don't have to check the return value here
    return ndev;
}

PYGORPHO_API int pyGetDeviceName(int device, char *nameBuffer)
{
    hipDeviceProp_t props;
    if (hipGetDeviceProperties(&props, device) != hipSuccess) {
        return ERR_BAD_CUDA_DEVICE;
    }

    std::strncpy(nameBuffer, props.name, 256);

    return 0;
}

PYGORPHO_API int pyFlatBallApproxStrel(int *lineSteps, int *lineLens, int radius, int typeCode)
{
    // NOTE: It is assumed that lineSteps and lineLens point to allocated memory block of adequate size
    std::vector<gpho::LineSeg> lines;
    TRY_OR_RETURN_ERROR(
        lines = gpho::flatBallApprox(radius, getApproxTypeFromCode(typeCode));
    )
    int i = 0;
    for (const auto& line : lines) {
        lineSteps[i * 3 + 0] = line.step.x;
        lineSteps[i * 3 + 1] = line.step.y;
        lineSteps[i * 3 + 2] = line.step.z;
        lineLens[i] = line.length;
        ++i;
    }
    return SUCCESS;
}

PYGORPHO_API int pyGenDilateErode(void *res, const void *vol, const void *strel,
    int volX, int volY, int volZ, int strelX, int strelY, int strelZ, int type, int op,
    int blockX, int blockY, int blockZ)
{
    if (pyGetDeviceCount() < 1) return ERR_NO_AVAILABLE_CUDA_DEVICE;
    int3 volSize = make_int3(volX, volY, volZ);
    int3 strelSize = make_int3(strelX, strelY, strelZ);
    int3 blockSize = make_int3(blockX, blockY, blockZ);
    TRY_OR_RETURN_ERROR(
        typeDispatch(type, doGenDilateErode, res, vol, strel, volSize, strelSize, op, blockSize);
    )
    return SUCCESS;
}

PYGORPHO_API int pyFlatDilateErode(void *res, const void *vol, const bool *strel,
    int volX, int volY, int volZ, int strelX, int strelY, int strelZ, int type, int op,
    int blockX, int blockY, int blockZ)
{
    if (pyGetDeviceCount() < 1) return ERR_NO_AVAILABLE_CUDA_DEVICE;
    int3 volSize = make_int3(volX, volY, volZ);
    int3 strelSize = make_int3(strelX, strelY, strelZ);
    int3 blockSize = make_int3(blockX, blockY, blockZ);
    TRY_OR_RETURN_ERROR(
        typeDispatch(type, doFlatDilateErode, res, vol, strel, volSize, strelSize, op, blockSize);
    )
    return SUCCESS;
}

PYGORPHO_API int pyFlatLinearDilateErode(void *res, const void *vol, const int *lineSteps,
    const int *lineLens, int volX, int volY, int volZ, int numLines, int type, int op,
    int blockX, int blockY, int blockZ)
{
    if (pyGetDeviceCount() < 1) return ERR_NO_AVAILABLE_CUDA_DEVICE;
    int3 volSize = make_int3(volX, volY, volZ);
    int3 blockSize = make_int3(blockX, blockY, blockZ);
    TRY_OR_RETURN_ERROR(
        typeDispatch(type, doFlatLinearDilateErode, res, vol, lineSteps,
            lineLens, volSize, numLines, op, blockSize);
    )
    return SUCCESS;
}

#ifdef __cplusplus
}
#endif